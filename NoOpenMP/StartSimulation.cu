#include "hip/hip_runtime.h"
#include "StartSimulation.h"
#include "Body.h"
#include "QuadNode.h"
#include <GL/glut.h>
#include <omp.h>

#include "cuPrintf.cu"

#include <string>
#include <time.h> 
#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand */      
#include <iostream>
#include <vector>

using namespace std;

#define TIMETORUN 50


//// nvcc hello-world.cu -L /usr/local/cuda/lib -lcudart -o hello-world
//CUDA
const int N = 16; 
const int blocksize = 16; 

__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
	int myid = blockIdx.x;
	cuPrintf("hello%d\n", myid);
}

__global__ 
void ComputeForce(Body* Bodies) 
{
	//a[threadIdx.x] += b[threadIdx.x];
	int myid = blockIdx.x;
	int j;
	double x_dist,y_dist;
	double r_Squared;
	double dist;
	double Force;
	cuPrintf("hello%d\n", myid);


	Bodies[myid].fx = 0;
	Bodies[myid].fy = 0;

	for(j = 0 ; j< gridDim.x; j++){

				if(myid!=j){
					
					    x_dist = Bodies[myid].x - Bodies[j].x;
						y_dist = Bodies[myid].y - Bodies[j].y;
						r_Squared = (x_dist*x_dist)  + (y_dist*y_dist);
						dist = sqrt(r_Squared);

						if(dist > 10){

							Force = (Bodies[myid].mass * Bodies[j].mass )/ (dist *dist * gridDim.x * gridDim.x * gridDim.x);
							Bodies[myid].fx -= Force * x_dist ;/// dist;
							Bodies[myid].fy -= Force * y_dist ;/// dist;

						}
				}
	}


 __syncthreads();

	for(j = 0 ; j< gridDim.x; j++){
		Bodies[myid].vx += Bodies[myid].fx / Bodies[myid].mass;
        Bodies[myid].vy += Bodies[myid].fy / Bodies[myid].mass;

        Bodies[myid].x += Bodies[myid].vx;
        Bodies[myid].y += Bodies[myid].vy;
	}	
}


	

StartSimulation::StartSimulation(int count,int x, int y){
	//cuda
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("%s\n", a);




	//Body *myBodies[256];
	this->numOfBodies = count;

	int i;
	double ina,inb,inc;
	time_t seconds;
	time(&seconds);
	srand((unsigned int) seconds);
	
	//temporary
	Body *myList [count];

	//QuadTree
	this->mytree = new QuadNode(0,x,0,y);

	//this->myBodies = myList[0];
	myBodies = new Body[count];

	for(i = 0; i < count; i++){

		ina = rand()%x;//x
		inb = rand()%y;//y
		inc = rand()%1000;//mass

		myList[i] = new Body(ina,inb,0.0,0.0,inc);
	
		this->myBodies[i] = *myList[i];

		//delete *myList;	

	}
	//this->myBodies = *myList;
	
	
	printf("\nInitialize Bodies done!\n");


	//start timestep
	
	for(i = 0; i < TIMETORUN ; i++){



	}




}

void StartSimulation::run(int choice){
	
	if(choice == 0){
	int i,j;
		
		for(i = 0 ; i < this->numOfBodies ; i++){

			this->myBodies[i].resetForce();
		
			for(j = 0 ; j< this->numOfBodies ; j++){

				if(i!=j){
					this->myBodies[i].addForce(this->myBodies[j]);
				}
			}
			this->myBodies[i].update(1);
		}

	
	}

	else if(choice == 1){
		int count = this->numOfBodies;
		this->convertedBodies = &myBodies;
		TreeRun(count, this->myBodies, this->mytree);
		
	}
	//OpenMP
	else if(choice == 2){
		int count = this->numOfBodies;
		this->convertedBodies = &myBodies;
		TreeRunOpenMP(count, this->myBodies, this->mytree);
	}

	//brute force CUDA
	else if(choice == 3){

		//char a[N] = "Hello \0\0\0\0\0\0";
		//int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

		Body *CBody;
		const int CSize = this->numOfBodies * sizeof(Body);

		hipMalloc( (void**)&CBody, CSize);

		hipMemcpy(CBody, this->myBodies,CSize, hipMemcpyHostToDevice);

		dim3 dimBlock(1,1);
		dim3 dimGrid(this->numOfBodies,1);

		ComputeForce<<<dimGrid,dimBlock>>>(CBody);

		hipMemcpy(this->myBodies,CBody,CSize, hipMemcpyDeviceToHost);


		hipFree(CBody);
		/*
		printf("%s", a);

		hipMalloc( (void**)&ad, csize ); 
		hipMalloc( (void**)&bd, isize ); 
		hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
		hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
		dim3 dimBlock( blocksize, 1 );
		dim3 dimGrid( 1, 1 );
		hello<<<dimGrid, dimBlock>>>(ad, bd);
		hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
		hipFree( ad );
		hipFree( bd );
	
		printf("%s\n", a);
		*/
	

	}



}

Body StartSimulation::GetBody(int i){
	return this->myBodies[i];
}

void StartSimulation::TreeRun(int count, Body *myList, QuadNode *tree)
{
	tree->clearNode();
	//this->mytree = new QuadNode(0,x,0,y);
	Body *tempList;
	for(int i = 0 ; i < count ; i++){
		tempList = &myList[i];
		tree->addBody(tempList);
		// printf("\nafter insert");
		// myList[i]->toString();			
	}


	for(int i = 0 ; i < count ; i++){
		myList[i].resetForce();

		myList[i].calcForce(tree);
		//printf("\nafter calc");
		//myList[i]->toString();
		myList[i].calcPosition(10);
		//myList[i].toString();	
	}


	
}
void StartSimulation::TreeRunOpenMP(int count, Body *myList, QuadNode *tree)
{
	tree->clearNode();
	//this->mytree = new QuadNode(0,x,0,y);
	Body *tempList;
	for(int i = 0 ; i < count ; i++){
		tempList = &myList[i];
		tree->addBody(tempList);
		// printf("\nafter insert");
		// myList[i]->toString();			
	}

//omp_set_num_threads(4);

//#pragma omp parallel shared(myList) private(i)
//#pragma omp for

	for(int i = 0 ; i < count ; i++){
		myList[i].resetForce();

		myList[i].calcForce(tree);
		//printf("\nafter calc");
		//myList[i]->toString();
		myList[i].calcPosition(10);
		//myList[i].toString();	
	}


	
}
