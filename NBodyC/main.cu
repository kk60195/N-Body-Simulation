#include "hip/hip_runtime.h"
//gcc -O1 -fopenmp -o main main.c -lrt -lm


#include <stdio.h> //fopen/fclose/printf/gets/puts
#include <stdlib.h> //variable types defined size_t/malloc/atoi/rand/abs
#include <time.h> //struct timspec/clock_gettime/CLOCK_PROCESS_CPUTIME_TIM
#include <math.h>
#include <pthread.h> // pthreads
#include <omp.h> // openmp

#include "cuPrintf.cu" // cuda print
#include "cuPrintf.cuh" //cuda print


#include <GL/glut.h> //openGL
//#include <sstream>
//#include <iomanip>


#define GIG 1000000000
#define CPG 3.168           // Cycles per GHz -- Adjust to your computer

#define YMAX 10000
#define XMAX 10000
#define CORRMIN -1E4 // display min
#define CORRMAX  1E4 // display max
#define MASSMAX 500
#define TIMESTEP 1
#define POINTSIZE 4.0

#define BODYPROPCOUNT 7 //x,y,m,vx,vy,fx,fy
#define GRAVITYCONST 6E-11
#define RADIUSINVISIBLE 100 //two planets dont see each other when they are within this distance

const int BLOCKSIZE = 1024; 

//openGL Constants
const int   SCREEN_WIDTH    = 400;
const int   SCREEN_HEIGHT   = 300;
const float CAMERA_DISTANCE = 10.0f;
const int   TEXT_WIDTH      = 80;
const int   TEXT_HEIGHT     = 130;

typedef double data_t;



//point structure used to render points in open GL
typedef struct Point
{
    float x, y;
    unsigned char r, g, b, a;
} Point_rec, *Point_ptr;

//holds points
typedef struct Points
{
Point_ptr data;
data_t len; // Total size of the vector
data_t current; //Number of vectors in it at present
} Points_rec, *Points_ptr;


//Text GL
void *font = GLUT_BITMAP_TIMES_ROMAN_24;

//strudture to hold galaxies
typedef struct {
	long int len;
	data_t *Bodies;
} Body_rec, *Body_ptr;

/* NUMBER of bytes in a vector */
#define VBYTES 16

/* Number o elements in a vector 
#define VSIZE VBYTES/sizeof(data_t)
typedef data_t Body_t _attribute_((vector_size(VBYTES)));
typedef union { 
	Body_t v;
	data_t d[VSIZE];

}pack_t;
*/


//Global Variables
Body_ptr Galaxy;
Points_ptr points;
double resultTotal;
int rounds;
int algorithmChoice;
int ManualNumBody;
/**********************************************************************/

//GPU

__global__ 
void ComputeForce(data_t *Bodies, long int len) 
{
  //a[threadIdx.x] += b[threadIdx.x];
  int myid = blockIdx.x * blockDim.x + threadIdx.x;
  int j;
  double x_dist,y_dist;
  double r_Squared;
  double dist;
  double Force;
  //cuPrintf("hello%d\n", myid);


  Bodies[myid + 5 *len] = 0;
  Bodies[myid + 6 *len] = 0;

  for(j = 0 ; j< len; j++){

        if(myid!=j){
          
            x_dist = Bodies[myid] - Bodies[j];
            y_dist = Bodies[myid + len] - Bodies[j + len];
            r_Squared = (x_dist*x_dist)  + (y_dist*y_dist);
            dist = sqrt(r_Squared);

            if(dist > RADIUSINVISIBLE){

              Force = (Bodies[myid + 2 * len] * Bodies[j + 2 * len] )/ (dist *dist * len /2);
              Bodies[myid + 5 * len] -= Force * x_dist ;/// dist;
              Bodies[myid + 6 * len] -= Force * y_dist ;/// dist;

            }
        }
  }


 //__syncthreads();

  for(j = 0 ; j< gridDim.x; j++){
        Bodies[myid + 3*len] += Bodies[myid + 5 * len] * TIMESTEP/ Bodies[myid + 2 * len];
        Bodies[myid + 4*len] += Bodies[myid + 6 * len] * TIMESTEP/ Bodies[myid + 2 * len];

         Bodies[myid] += Bodies[myid + 3 * len] * TIMESTEP;
         Bodies[myid + len] += Bodies[myid + 4 * len] * TIMESTEP;
     
  } 
}




/******************************************************************/
main(int argc, char *argv[]){

//iterative variable
int i;

//variable declare


//function declares
//struct timespec diff(struct timespec start, struct timespec end);
//struct timespec time1,time2,Diff;
Body_ptr new_galaxy(long int len);
Points_ptr new_points(long int len);
void display();
void reshape(int w, int h);


//calculation functions
void BruteForce(Body_ptr G);



//parse ArgC
if (argc > 1) 
    algorithmChoice = atoi( argv[1] );
    else algorithmChoice = 0;

if(argc >= 2)
    ManualNumBody = atoi( argv[2] );
    else ManualNumBody = 200;
    

//create galaxy
Galaxy = new_galaxy(ManualNumBody);

//debug printing
// for(i = 0 ; i < BODYPROPCOUNT; i++){
// 	for(j = 0 ; j < ManualNumBody; j++){
// 		printf("%.1f\t",Galaxy->Bodies[i*ManualNumBody + j]);
// 	}
// 	printf("\n");
// }


if(false){ // non cuda runs
if( algorithmChoice == 0){

//brute force
	//clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
    BruteForce(Galaxy);
    //clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
    //Diff = diff(time1,time2);
    //printf("\nTime: %ld\n", (long int)((double)(GIG * Diff.tv_sec + Diff.tv_nsec)));
}

//bruteforce gpu
if(algorithmChoice == 1){

    data_t *CBody;
    const int CSize = Galaxy->len * BODYPROPCOUNT * sizeof(Body_rec);

    hipMalloc( (void**)&CBody, CSize);

    hipMemcpy(CBody, Galaxy->Bodies,CSize, hipMemcpyHostToDevice);

    dim3 dimBlock(1,1);
    dim3 dimGrid(ManualNumBody,1);

    ComputeForce<<<dimGrid,dimBlock>>>(CBody,Galaxy->len);

    hipMemcpy(Galaxy->Bodies,CBody,CSize, hipMemcpyDeviceToHost);


    hipFree(CBody);
}

}

//Create Points to show

points = new_points(ManualNumBody);

//open GL
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DEPTH | GLUT_DOUBLE);

    glutInitWindowSize(XMAX,YMAX);
    glutCreateWindow("Random Points");

    glutDisplayFunc(display);
    glutReshapeFunc(reshape);

//initate a set of points

    for( i = 0; i < ManualNumBody; i++ )
    {
        
        points->data[i].x = Galaxy->Bodies[i];
        points->data[i].y = Galaxy->Bodies[i + ManualNumBody];
        points->data[i].r = rand()%255;
        points->data[i].g = rand()%255;
        points->data[i].b = rand()%255;
        points->data[i].a = 255;
        
    }        

    glutIdleFunc(display);

    glutMainLoop();


}

/***********************************/


// create space
Body_ptr new_galaxy(long int len){
	//function declare
	int init_bodies_rand(Body_ptr v, long int len);
 	int success;

	//making the head object
	Body_ptr result = (Body_ptr)malloc(sizeof(Body_rec));
	//catch
	if(!result)return NULL;
	result->len = len;

	//Make the Galaxies x,y,vx,vy,fx,fy,m
	if(len > 0){
		data_t *Bodies = (data_t *) calloc(len*BODYPROPCOUNT, sizeof(data_t));
		if(!Bodies){
			free((void*) result);
			printf("\n COULDN'T ALLOCATE STORAGE \n", result->len);
			return NULL;
		}
		//assign object into head object
		result->Bodies = Bodies;

		//populate the bodies
		success = init_bodies_rand(result, result->len);

	}
	else result->Bodies = NULL;



	return result;
}

/************************************/
//calculate difference in timsepcs
struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}
//random function with seed
double fRand(double fMin, double fMax)
{
    double f = (double)random() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}

//fill bodies with x,y and m
int init_bodies_rand(Body_ptr G, long int len)
{
  long int i;
  double fRand(double fMin, double fMax);

  if (len > 0) {
    //G->len = len;
    for (i = 0; i < len; i++){
      G->Bodies[i] = (data_t)(fRand((double)(0),(double)(XMAX))); // x
  	  G->Bodies[1*len + i] = (data_t)(fRand((double)(0),(double)(YMAX))); //y
  	  G->Bodies[2*len + i] = (data_t)(fRand((double)(0),(double)(MASSMAX))); //m

  	}
    return 1;
  }
  else return 0;
}
/*****************************************************/
//points declare for OPENGL
Points_ptr new_points(long int len){
	

	//making the head object
	Points_ptr result = (Points_ptr)malloc(sizeof(Points_rec));
	//catch
	if(!result)return NULL;
	result->len = len;

	//Make the Galaxies x,y,vx,vy,fx,fy,m
	if(len > 0){
		Point_ptr points = (Point_ptr) calloc(len, sizeof(Point_rec));
		if(!points){
			free((void*) result);
			printf("\n COULDN'T ALLOCATE STORAGE \n", result->len);
			return NULL;
		}
		//assign object into head object
		result->data = points;

		result->current = 0;


	}
	else result->data = NULL;



	return result;


}
void crunch(long int len){
	//calculation functions

	void BruteForce(Body_ptr G);


     //struct timespec diff(struct timespec start, struct timespec end);
     struct timespec time1, time2,result;
     clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
    //run simulation
     if(algorithmChoice == 0){
	         BruteForce(Galaxy);
     }
     //run GPU
     if(algorithmChoice == 1){

         data_t *CBody;
         const int CSize = Galaxy->len * BODYPROPCOUNT * sizeof(data_t);

         hipMalloc( (void**)&CBody, CSize);

          hipMemcpy(CBody, Galaxy->Bodies,CSize, hipMemcpyHostToDevice);

          dim3 dimBlock(BLOCKSIZE,1);
          dim3 dimGrid(ManualNumBody/BLOCKSIZE,1);

          ComputeForce<<<dimGrid,dimBlock>>>(CBody,Galaxy->len);

          hipMemcpy(Galaxy->Bodies,CBody,CSize, hipMemcpyDeviceToHost);
          hipFree(CBody);

          hipMalloc( (void**)&CBody, CSize);

          hipMemcpy(CBody, Galaxy->Bodies,CSize, hipMemcpyHostToDevice);

          //dim3 dimBlock(ManualNumBody%1024,1);
          //dim3 dimGrid(1,1);

          ComputeForce<<<1,ManualNumBody%BLOCKSIZE>>>(CBody,Galaxy->len);

          hipMemcpy(Galaxy->Bodies,CBody,CSize, hipMemcpyDeviceToHost);
          hipFree(CBody);


     }


     clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
     result = diff(time1,time2);
     
     resultTotal = result.tv_sec * 1E3 +  result.tv_nsec * 1E-6;
     resultTotal = resultTotal;
     printf("CPU time:\t%.1f (msec)\n", resultTotal);
      //text
    
    //reset display points 
  	int i;
    for(  i = 0; i < len; i++ )
    {
        
        points->data[i].x = CORRMAX/2-XMAX + Galaxy->Bodies[i];
        points->data[i].y = CORRMAX/2-YMAX + Galaxy->Bodies[i + ManualNumBody];
        points->data[i].r = abs( (unsigned char)(Galaxy->Bodies[i + 2 * ManualNumBody]) % 255);
        points->data[i].g =  abs( (unsigned char)(Galaxy->Bodies[i + 2 * ManualNumBody]) % 255);
        points->data[i].b = 255;
        points->data[i].a = 255;

    

    }        

}


void setupGL(){

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(CORRMIN,CORRMAX,CORRMIN,CORRMAX, -1, 1);


    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    // draw
    glColor3ub( 255, 255, 255 );
    glEnableClientState( GL_VERTEX_ARRAY );
    glEnableClientState( GL_COLOR_ARRAY );

    //smooth
    glEnable( GL_POINT_SMOOTH );
    glEnable( GL_BLEND );
    
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    
    glPointSize( POINTSIZE );

}

void reshape(int w, int h)
{
    glViewport(0, 0, w, h);
}

void showInfo(double timing)
{
    // backup current model-view matrix
    glPushMatrix();                     // save current modelview matrix
    glLoadIdentity();                   // reset modelview matrix

    // set to 2D orthogonal projection
    glMatrixMode(GL_PROJECTION);        // switch to projection matrix
    glPushMatrix();                     // save current projection matrix
    glLoadIdentity();                   // reset projection matrix
    gluOrtho2D(0, XMAX, 0, YMAX); // set to orthogonal projection

    float color[4] = {1, 1, 1, 1};

    //stringstream ss;
    //ss << std::fixed << std::setprecision(3);

    //ss << "Timing " << timing <<  "Msec" << " \n #Obj: " << ManualNumBody << "\n Iter: " << rounds <<ends;
    //drawString(ss.str().c_str(), 1, 950, color, font);
    //ss.str("");


    // unset floating format
    //ss << std::resetiosflags(std::ios_base::fixed | std::ios_base::floatfield);

    // restore projection matrix
    glPopMatrix();                   // restore to previous projection matrix

    // restore modelview matrix
    glMatrixMode(GL_MODELVIEW);      // switch to modelview matrix
    glPopMatrix();                   // restore to previous modelview matrix
}

void display()
{


    //run simulation
     crunch(points->len);

     //refactor
     setupGL();
    
    //draw
    glVertexPointer( 2, GL_FLOAT, sizeof(Point_ptr), &(points->data[0].x) );
    glColorPointer( 4, GL_UNSIGNED_BYTE, sizeof(Point_ptr), &(points->data[0].r) );

    glDrawArrays( GL_POINTS, 0, points->len );

    glDisableClientState( GL_VERTEX_ARRAY );
    glDisableClientState( GL_COLOR_ARRAY );

     
    //display text
    //showInfo(resultTotal/rounds);
   
    glFlush(); // dont need flush because swap buffer has it intrinsically..used before for single buffer
    glutSwapBuffers();
    glutReshapeFunc(reshape);
}


/******************************************/
void BruteForce(Body_ptr G){

//iterators
int i,j,len;
len = G->len;

//reference placement
long int x,y,m,vx,vy,fx,fy;
x = 0;
y = len;
m = len * 2;
vx = len * 3;
vy = len * 4;
fx = len * 5;
fy = len * 6;

//calculation
double m1,m2,dy,dx,dt,dtsq,force;

data_t *Bodies = G->Bodies;

//compute forces
for(i = 0 ; i < len; i++){

	Bodies[i + fx] = 0; //reset fx
	Bodies[i + fy] = 0; //reset fy

	for( j = 0; j < len; j++){
		if(i!=j){

			m1 = Bodies[i + m];
			m2 = Bodies[j + m];
			dx = Bodies[i + x] - Bodies[j + x];
			dy = Bodies[i + y] - Bodies[j + y];
			dtsq = dx * dx + dy * dy;
			dt = sqrt(dtsq);
			force = (m1 * m2  * GRAVITYCONST * 6E11) / (dtsq);

			//Set force
      if(dt > 100){
		    Bodies[i + fx] += (dx/dt) * force; //set fx
		    Bodies[i + fy] += (dy/dt) * force; //set fy
      }
		}
	}

}

//apply forces
for( i = 0 ; i < len ; i++){


	Bodies[i + vx] += Bodies[i + fx] * TIMESTEP / Bodies[i + m]; //set vx += fx /m
	Bodies[i + vy] += Bodies[i + fy] * TIMESTEP / Bodies[i + m]; //set vy += fy /m

	Bodies[i + x] -= Bodies[i + vx] * TIMESTEP; //new x +=vx
	Bodies[i + y] -= Bodies[i + vy] * TIMESTEP; //new y +=vy
}


}