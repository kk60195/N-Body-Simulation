#include "hip/hip_runtime.h"
//gcc -O1 -fopenmp -o main main.c -lrt -lm


#include <stdio.h> //fopen/fclose/printf/gets/puts
#include <stdlib.h> //variable types defined size_t/malloc/atoi/rand/abs
#include <time.h> //struct timspec/clock_gettime/CLOCK_PROCESS_CPUTIME_TIM
#include <math.h>
#include <pthread.h> // pthreads
#include <omp.h> // openmp

#include "cuPrintf.cu" // cuda print
#include "cuPrintf.cuh" //cuda print


#include <GL/glut.h> //openGL
//#include <sstream>
//#include <iomanip>


#define GIG 1000000000
#define CPG 3.168           // Cycles per GHz -- Adjust to your computer

#define YMAX 10000
#define XMAX 10000
#define CORRMIN -1E4 // display min
#define CORRMAX  1E4 // display max
#define MASSMAX 500
#define TIMESTEP 1
#define POINTSIZE 2.0

#define BODYPROPCOUNT 7 //x,y,m,vx,vy,fx,fy
#define GRAVITYCONST 6E-11
#define RADIUSINVISIBLE 100 //two planets dont see each other when they are within this distance


#define TREELEVEL 6

#define TREEPROPCOUNT 11 //xmin, xmax, ymin, ymax, m, mx, my, child1, child2, child3, child4
#define THETA 0.5

const int BLOCKSIZE = 1024; 

//openGL Constants
const int   SCREEN_WIDTH    = 400;
const int   SCREEN_HEIGHT   = 300;
const float CAMERA_DISTANCE = 10.0f;
const int   TEXT_WIDTH      = 80;
const int   TEXT_HEIGHT     = 130;

typedef float data_t;



//point structure used to render points in open GL
typedef struct Point
{
    float x, y;
    unsigned char r, g, b, a;
} Point_rec, *Point_ptr;

//holds points
typedef struct Points
{
Point_ptr data;
data_t len; // Total size of the vector
data_t current; //Number of vectors in it at present
} Points_rec, *Points_ptr;


//Text GL
void *font = GLUT_BITMAP_TIMES_ROMAN_24;

//strudture to hold galaxies, include all the bodies
typedef struct {
	long int len;
	data_t *Bodies;
} Body_rec, *Body_ptr;

//structure to hold the tree, include all the tree nodes
typedef struct{
  long int numNode;
  data_t *Nodes;
} Node_rec, *Node_ptr;

/* NUMBER of bytes in a vector */
#define VBYTES 16



//Global Variables
Body_ptr Galaxy;
Points_ptr points;
float resultTotal;
int rounds;
int algorithmChoice;
int ManualNumBody;
Node_ptr MyTree;



/**********************************************************************/
//cuda bruteforce
__global__ 
void ComputeForce(data_t *Bodies, long int len) 
{

  
  //a[threadIdx.x] += b[threadIdx.x];
  int myid = blockIdx.x * blockDim.x + threadIdx.x;
  int j;
  float x_dist,y_dist;
  float r_Squared;
  float dist;
  float Force;
  //cuPrintf("hello%d\n", myid);
  if(myid < len){

    Bodies[myid + 5 *len] = 0;
    Bodies[myid + 6 *len] = 0;
    
    //for each body, calculate it force against all the other bodies
    for(j = 0 ; j< len; j++){

        if(myid!=j){
          
            x_dist = Bodies[myid] - Bodies[j];
            y_dist = Bodies[myid + len] - Bodies[j + len];
            r_Squared = (x_dist*x_dist)  + (y_dist*y_dist);
            dist = sqrt(r_Squared);

            if(dist > RADIUSINVISIBLE){

              Force = (Bodies[myid + 2 * len] * Bodies[j + 2 * len] )/ (dist *dist * len /2);
              Bodies[myid + 5 * len] -= Force * x_dist ;/// dist;
              Bodies[myid + 6 * len] -= Force * y_dist ;/// dist;

            }
        }
    }

    for(j = 0 ; j< gridDim.x; j++){
          Bodies[myid + 3*len] += Bodies[myid + 5 * len] * TIMESTEP/ Bodies[myid + 2 * len];
          Bodies[myid + 4*len] += Bodies[myid + 6 * len] * TIMESTEP/ Bodies[myid + 2 * len];

          Bodies[myid] += Bodies[myid + 3 * len] * TIMESTEP;
          Bodies[myid + len] += Bodies[myid + 4 * len] * TIMESTEP;
       
    } 
  }
}



/***********************************/
//cuda code for barnes-hut
__global__ 
void ComputeBarnes(data_t *Bodies ,long int len, data_t * Nodes , long int num, int num0) 
{


  int myid = blockIdx.x * blockDim.x + threadIdx.x;
  int i, j,k;
  float x_dist,y_dist,dtsq,dist,m1,m2;
  float force;

//for each body, calculate it's force against all the lowest level tree nodes
  if(myid < len){

    Bodies[myid + 5 *len] = 0;
    Bodies[myid + 6 *len] = 0;
    
    for (j = num0; j < num; j++ ){
      x_dist = Bodies[myid] - Nodes[j + num*5];
      y_dist = Bodies[myid + len] - Nodes[j + num*6];
      dtsq = x_dist*x_dist + y_dist*y_dist;
      dist = sqrt(dtsq);
      m1 = Bodies[myid + len*2];
      m2 = Nodes[j + num*4];
      if(dist > 100){
        force = (m1 * m2  * GRAVITYCONST * 6E9) / (dtsq);
         
        Bodies[myid + len*5] +=  (-x_dist/dist) * force ;
        Bodies[myid + len*6] +=  (-y_dist/dist) * force ;
      }
    }

    Bodies[myid + 3*len] += Bodies[myid + 5 * len] * TIMESTEP/ Bodies[myid + 2 * len];
    Bodies[myid + 4*len] += Bodies[myid + 6 * len] * TIMESTEP/ Bodies[myid + 2 * len];

    Bodies[myid] += Bodies[myid + 3 * len] * TIMESTEP;
    Bodies[myid + len] += Bodies[myid + 4 * len] * TIMESTEP;
  }

}


/******************************************************************/
main(int argc, char *argv[]){

  //iterative variable
  int i, j;


  Body_ptr new_galaxy(long int len);
  Points_ptr new_points(long int len);
  void display();
  void reshape(int w, int h);
  Node_ptr new_tree();


  //calculation functions
  void BruteForce(Body_ptr G);



  //parse ArgC
  if (argc > 1) 
    algorithmChoice = atoi( argv[1] );
  else algorithmChoice = 0;

  if(argc >= 2)
    ManualNumBody = atoi( argv[2] );
  else ManualNumBody = 200;
      

  //create galaxy
  Galaxy = new_galaxy(ManualNumBody);

  MyTree = new_tree();





  //needed
  points = new_points(ManualNumBody);

  //open GL
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DEPTH | GLUT_DOUBLE);

  glutInitWindowSize(XMAX,YMAX);
  glutCreateWindow("Random Points");

  glutDisplayFunc(display);
  glutReshapeFunc(reshape);

//initate a set of points

  for( i = 0; i < ManualNumBody; ++i )
  {
      
      points->data[i].x = Galaxy->Bodies[i];
      points->data[i].y = Galaxy->Bodies[i + ManualNumBody];
      points->data[i].r = rand()%255;
      points->data[i].g = rand()%255;
      points->data[i].b = rand()%255;
      points->data[i].a = 255;
      
  }        

  glutIdleFunc(display);

  glutMainLoop();


}

/***********************************/


// create space
Body_ptr new_galaxy(long int len){
	//function declare
	int init_bodies_rand(Body_ptr v, long int len);
 	int success;

	//making the head object
	Body_ptr result = (Body_ptr)malloc(sizeof(Body_rec));
	//catch
	if(!result)return NULL;
	result->len = len;

	//Make the Galaxies x,y,vx,vy,fx,fy,m
	if(len > 0){
		data_t *Bodies = (data_t *) calloc(len*BODYPROPCOUNT, sizeof(data_t));
		if(!Bodies){
			free((void*) result);
			printf("\n COULDN'T ALLOCATE STORAGE \n", result->len);
			return NULL;
		}
		//assign object into head object
		result->Bodies = Bodies;

		//populate the bodies
		success = init_bodies_rand(result, result->len);

	}
	else result->Bodies = NULL;



	return result;
}


/************************************/
//create tree 
Node_ptr new_tree(){
  int init_tree(Node_ptr T, int curIndex, int curLevel);//initialize xmin xmax ymin ymax
  long int num = (pow(4,TREELEVEL) - 1)/3;
  int success;

  Node_ptr nodept = (Node_ptr)malloc(sizeof(Node_rec));
  //catch
  if(!nodept)return NULL;
   nodept->numNode = num;

  //make the tree
  if(nodept->numNode > 0){
    data_t *Nodes = (data_t *)calloc(nodept->numNode*TREEPROPCOUNT, sizeof(data_t));
    if(!Nodes){
      free((void*) nodept);
      printf("\n COULDN'T ALLOCATE STORAGE \n", nodept->numNode);
      return NULL;
    }
    //
    nodept->Nodes = Nodes;
    success = init_tree(nodept, 0, 1);
  }else{
    nodept->Nodes = NULL;
  }

  return nodept;
}

/************************************/
//calculate difference in timsepcs
struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}
//random function with seed
float fRand(float fMin, float fMax)
{
    float f = (float)random() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}


/********/
//fill bodies with x,y and m
int init_bodies_rand(Body_ptr G, long int len)
{
  long int i;
  float fRand(float fMin, float fMax);

  if (len > 0) {
    //G->len = len;
    for (i = 0; i < len; i++){
      G->Bodies[i] = (data_t)(fRand((float)(0),(float)(XMAX))); // x
  	  G->Bodies[1*len + i] = (data_t)(fRand((float)(0),(float)(YMAX))); //y
  	  G->Bodies[2*len + i] = (data_t)(fRand((float)(0),(float)(MASSMAX))); //m

  	}
    return 1;
  }
  else return 0;
}




/*************************************/
//reset the tree
void reset_tree(){
  int i,j;
  int num = MyTree-> numNode;

  for(i = 4 ; i < 7; i++){                //
    for(j = 0; j < num; j++){
      MyTree->Nodes[num*i + j] = 0;
    }
  }
}



/************************************/
//init treenodes
int init_tree(Node_ptr T, int curIndex, int curLevel)
{
  long int i,j;
  long int temp1,temp2,temp3,temp4;
  long int num = T->numNode;


  if (num > 0){
//for root node
    if(curIndex == 0){

      T->Nodes[curIndex] = 0;
      T->Nodes[curIndex + 1*num] = XMAX;
      T->Nodes[curIndex + 2*num] = 0;
      T->Nodes[curIndex + 3*num] = YMAX;

      T->Nodes[curIndex + 7*num] = curIndex*4+1;
      T->Nodes[curIndex + 8*num] = curIndex*4+2;
      T->Nodes[curIndex + 9*num] = curIndex*4+3;
      T->Nodes[curIndex + 10*num] = curIndex*4+4; 
      init_tree(T, curIndex*4+1, curLevel+1);
      init_tree(T, curIndex*4+2, curLevel+1);
      init_tree(T, curIndex*4+3, curLevel+1);
      init_tree(T, curIndex*4+4, curLevel+1);

    }else{//not rootnode
      //get mid values of parent's region
      temp1 = (curIndex-1)/4;//parent's index
      temp2 = curIndex-1 - 4*temp1;// figure out which child is the current node to it's parent
      temp3 = (T->Nodes[temp1] + T->Nodes[temp1 + 1*num])/2;
      temp4 = (T->Nodes[temp1 + 2*num] + T->Nodes[temp1 + 3*num])/2;
      if(temp2==0){// bottom left
        T->Nodes[curIndex] = T->Nodes[temp1];
        T->Nodes[curIndex + 1*num] = temp3;
        T->Nodes[curIndex + 2*num] = T->Nodes[temp1 + 2*num];
        T->Nodes[curIndex + 3*num] = temp4;
      }else if(temp2==1){//bottom right
        T->Nodes[curIndex] = temp3;
        T->Nodes[curIndex + 1*num] = T->Nodes[temp1 + 1*num];
        T->Nodes[curIndex + 2*num] = T->Nodes[temp1 + 2*num];
        T->Nodes[curIndex + 3*num] = temp4;
      }else if(temp2==2){//top left
        T->Nodes[curIndex] = T->Nodes[temp1];
        T->Nodes[curIndex + 1*num] = temp3;
        T->Nodes[curIndex + 2*num] = temp4;
        T->Nodes[curIndex + 3*num] = T->Nodes[temp1 + 3*num];
      }else if(temp2==3){//top right
        T->Nodes[curIndex] = temp3;
        T->Nodes[curIndex + 1*num] = T->Nodes[temp1 + 1*num];
        T->Nodes[curIndex + 2*num] = temp4;
        T->Nodes[curIndex + 3*num] = T->Nodes[temp1 + 3*num];
      }else return 0;

      //for nodes on higher levels, specify their children
      if(curLevel<TREELEVEL){
        T->Nodes[curIndex + 7*num] = curIndex*4+1;
        T->Nodes[curIndex + 8*num] = curIndex*4+2;
        T->Nodes[curIndex + 9*num] = curIndex*4+3;
        T->Nodes[curIndex + 10*num] = curIndex*4+4;
        init_tree(T, curIndex*4+1, curLevel+1);
        init_tree(T, curIndex*4+2, curLevel+1);
        init_tree(T, curIndex*4+3, curLevel+1);
        init_tree(T, curIndex*4+4, curLevel+1);
      }else{//if on the last level
        T->Nodes[curIndex + 7*num] = -1;
        T->Nodes[curIndex + 8*num] = -1;
        T->Nodes[curIndex + 9*num] = -1;
        T->Nodes[curIndex + 10*num] = -1;
      }

    }
    return 1;

    
  }
  else return 0;
}

/*****************************************************/
//points declare for OPENGL
Points_ptr new_points(long int len){
	

	//making the head object
	Points_ptr result = (Points_ptr)malloc(sizeof(Points_rec));
	//catch
	if(!result)return NULL;
	result->len = len;

	//Make the Galaxies x,y,vx,vy,fx,fy,m
	if(len > 0){
		Point_ptr points = (Point_ptr) calloc(len, sizeof(Point_rec));
		if(!points){
			free((void*) result);
			printf("\n COULDN'T ALLOCATE STORAGE \n", result->len);
			return NULL;
		}
		//assign object into head object
		result->data = points;

		result->current = 0;


	}
	else result->data = NULL;



	return result;


}


/***************************************/
//perform one cycle calculations
void crunch(long int len){
	//calculation functions

	void BruteForce(Body_ptr G);
  void quadTree();
  void insertBody(int bodyIndex, int curIndex, int curLevel);

  //struct timespec diff(struct timespec start, struct timespec end);
  struct timespec time1, time2,result;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);

//run simulation
//cpu brute-force
  if(algorithmChoice == 0){
    BruteForce(Galaxy);
  }

//cuda brute-force
  if(algorithmChoice == 1){

    data_t *CBody;
    const int CSize = Galaxy->len * BODYPROPCOUNT * sizeof(data_t);

    hipMalloc( (void**)&CBody, CSize);

    hipMemcpy(CBody, Galaxy->Bodies,CSize, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCKSIZE,1);
    dim3 dimGrid(ManualNumBody/BLOCKSIZE+1,1);

    ComputeForce<<<dimGrid,dimBlock>>>(CBody,Galaxy->len);

    hipMemcpy(Galaxy->Bodies,CBody,CSize, hipMemcpyDeviceToHost);
    hipFree(CBody);
  }
//cpu barnes-hut
  if(algorithmChoice == 2){
    quadTree();
  }

//cuda barnes-hut
  if(algorithmChoice == 3){
    void insertBody(int bodyIndex, int curIndex, int curLevel);
    void reset_tree();
    reset_tree();

    int i,j,len,num,num0;
    len = Galaxy->len;
    num = MyTree->numNode;

    long int bx,by,bm,bvx,bvy,bfx,bfy;

    
    bx = 0;
    by = len;
    bm = len * 2;
    bvx = len * 3;
    bvy = len * 4;
    bfx = len * 5;
    bfy = len * 6;
    num0 = (pow(4, TREELEVEL-1) - 1)/3;


    for(i = 0; i < len; i++){
      insertBody(i,0,1);
    }

    data_t *CBody;
    data_t *CNode;

    const int CBodySize = Galaxy->len * BODYPROPCOUNT * sizeof(data_t);
    const int CNodeSize = MyTree->numNode * TREEPROPCOUNT * sizeof(data_t);

    hipMalloc( (void**)&CBody, CBodySize);
    hipMalloc((void**)&CNode, CNodeSize);

    hipMemcpy(CBody, Galaxy->Bodies,CBodySize, hipMemcpyHostToDevice);
    hipMemcpy(CNode, MyTree->Nodes,CNodeSize, hipMemcpyHostToDevice); 

    dim3 dimBlock(BLOCKSIZE,1);
    dim3 dimGrid(ManualNumBody/BLOCKSIZE+1,1);

    ComputeBarnes<<<dimGrid,dimBlock>>>(CBody,Galaxy->len,CNode,MyTree->numNode,num0);

    hipMemcpy(Galaxy->Bodies,CBody,CBodySize, hipMemcpyDeviceToHost);
    hipMemcpy(MyTree->Nodes,CNode,CNodeSize, hipMemcpyDeviceToHost);
    hipFree(CBody);
    hipFree(CNode);
  }

  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
  result = diff(time1,time2);
   
  resultTotal = result.tv_sec * 1E3 +  result.tv_nsec * 1E-6;
  resultTotal = resultTotal;
  printf("CPU time:\t%.1f (msec)\n", resultTotal);
    //text
  
  //reset display points 
	int i;
  for(  i = 0; i < len; ++i )
  {        
    points->data[i].x = CORRMAX/2-XMAX + Galaxy->Bodies[i];
    points->data[i].y = CORRMAX/2-YMAX + Galaxy->Bodies[i + ManualNumBody];
    points->data[i].r = abs( (unsigned char)(Galaxy->Bodies[i + 2 * ManualNumBody]) % 255);
    points->data[i].g =  abs( (unsigned char)(Galaxy->Bodies[i + 2 * ManualNumBody]) % 255);
    points->data[i].b = 255;
    points->data[i].a = 255;
  }        

}


void setupGL(){

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(CORRMIN,CORRMAX,CORRMIN,CORRMAX, -1, 1);


    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    // draw
    glColor3ub( 255, 255, 255 );
    glEnableClientState( GL_VERTEX_ARRAY );
    glEnableClientState( GL_COLOR_ARRAY );

    //smooth
    glEnable( GL_POINT_SMOOTH );
    glEnable( GL_BLEND );
    
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    
    glPointSize( POINTSIZE );

}

void reshape(int w, int h)
{
    glViewport(0, 0, w, h);
}

void showInfo(float timing)
{
    // backup current model-view matrix
    glPushMatrix();                     // save current modelview matrix
    glLoadIdentity();                   // reset modelview matrix

    // set to 2D orthogonal projection
    glMatrixMode(GL_PROJECTION);        // switch to projection matrix
    glPushMatrix();                     // save current projection matrix
    glLoadIdentity();                   // reset projection matrix
    gluOrtho2D(0, XMAX, 0, YMAX); // set to orthogonal projection

    float color[4] = {1, 1, 1, 1};


    glPopMatrix();                   // restore to previous projection matrix

    // restore modelview matrix
    glMatrixMode(GL_MODELVIEW);      // switch to modelview matrix
    glPopMatrix();                   // restore to previous modelview matrix
}

void display()
{
    //run simulation
     crunch(points->len);

     //refactor
     setupGL();
    
    //draw
    glVertexPointer( 2, GL_FLOAT, sizeof(Point_rec), &(points->data[0].x) );
    glColorPointer( 4, GL_UNSIGNED_BYTE, sizeof(Point_rec), &(points->data[0].r) );

    glDrawArrays( GL_POINTS, 0, points->len );

    glDisableClientState( GL_VERTEX_ARRAY );
    glDisableClientState( GL_COLOR_ARRAY );
     
    //display text   
    glFlush(); // dont need flush because swap buffer has it intrinsically..used before for single buffer
    glutSwapBuffers();
    glutReshapeFunc(reshape);
}


/******************************************/
//the calculate force func for cpu bruteforce method
void BruteForce(Body_ptr G){

  //iterators
  int i,j,len;
  len = G->len;

  //reference placement
  long int x,y,m,vx,vy,fx,fy;
  x = 0;
  y = len;
  m = len * 2;
  vx = len * 3;
  vy = len * 4;
  fx = len * 5;
  fy = len * 6;

  //calculation
  float m1,m2,dy,dx,dt,dtsq,force;

  data_t *Bodies = G->Bodies;

  //compute forces
  for(i = 0 ; i < len; i++){

  	Bodies[i + fx] = 0; //reset fx
  	Bodies[i + fy] = 0; //reset fy

  	for( j = 0; j < len; j++){
  		if(i!=j){

  			m1 = Bodies[i + m];
  			m2 = Bodies[j + m];
  			dx = Bodies[i + x] - Bodies[j + x];
  			dy = Bodies[i + y] - Bodies[j + y];
  			dtsq = dx * dx + dy * dy;
  			dt = sqrt(dtsq);
  			force = (m1 * m2  * GRAVITYCONST * 6E11) / (dtsq);

  			//Set force
        if(dt > 100){
  		    Bodies[i + fx] += (dx/dt) * force; //set fx
  		    Bodies[i + fy] += (dy/dt) * force; //set fy
        }
  		}
  	}

  }
  //apply forces
  for( i = 0 ; i < len ; i++){


  	Bodies[i + vx] += Bodies[i + fx] * TIMESTEP / Bodies[i + m]; //set vx += fx /m
  	Bodies[i + vy] += Bodies[i + fy] * TIMESTEP / Bodies[i + m]; //set vy += fy /m

  	Bodies[i + x] -= Bodies[i + vx] * TIMESTEP; //new x +=vx
  	Bodies[i + y] -= Bodies[i + vy] * TIMESTEP; //new y +=vy
  }
}

/************************************/
//insert bodies into tree
void insertBody(int bodyIndex, int curIndex, int curLevel){
  
  int i,j,len,num,key;
  long int bx,by,bm,bvx,bvy,bfx,bfy;
  long int nminx,nmaxx,nminy,nmaxy,nm,nmx,nmy,nch1,nch2,nch3,nch4;
  long int xmi,xma,ymi,yma;

  len = Galaxy->len;
  num = MyTree->numNode;
//parameters for bodies
  bx = 0;
  by = len;
  bm = len * 2;
  bvx = len * 3;
  bvy = len * 4;
  bfx = len * 5;
  bfy = len * 6;
//parameters for treenodes
  nminx = 0;
  nmaxx = num;
  nminy = num*2;
  nmaxy = num*3;
  nm = num*4;
  nmx = num*5;
  nmy = num*6;
  nch1 = num*7;
  nch2 = num*8;
  nch3 = num*9;
  nch3 = num*10;

  xmi = MyTree->Nodes[curIndex + nminx];
  xma = MyTree->Nodes[curIndex + nmaxx];
  ymi = MyTree->Nodes[curIndex + nminy];
  yma = MyTree->Nodes[curIndex + nmaxy];

//here we try to still insert the bodies that ran out of the region of the tree into root node and we enlarge the mass of root node to attract the bodies
  if(Galaxy->Bodies[bodyIndex + bx] < xmi || Galaxy->Bodies[bodyIndex + bx] > xma || Galaxy->Bodies[bodyIndex + by] < ymi || Galaxy->Bodies[bodyIndex + by] > yma){

    if(curLevel == 1){
        MyTree->Nodes[curIndex + nmx] = (MyTree->Nodes[curIndex + nmx] * MyTree->Nodes[curIndex + nm] + Galaxy->Bodies[bodyIndex + bx] * Galaxy->Bodies[bodyIndex + bm])/(MyTree->Nodes[curIndex + nm] + Galaxy->Bodies[bodyIndex + bm]);
        MyTree->Nodes[curIndex + nmy] = (MyTree->Nodes[curIndex + nmy] * MyTree->Nodes[curIndex + nm] + Galaxy->Bodies[bodyIndex + by] * Galaxy->Bodies[bodyIndex + bm])/(MyTree->Nodes[curIndex + nm] + Galaxy->Bodies[bodyIndex + bm]);
         MyTree->Nodes[curIndex + nm] += Galaxy->Bodies[bodyIndex + bm] * 200;

    }


    return;
  }

  //calculate the centre of mass and increase the mass of node 
  MyTree->Nodes[curIndex + nmx] = (MyTree->Nodes[curIndex + nmx] * MyTree->Nodes[curIndex + nm] + Galaxy->Bodies[bodyIndex + bx] * Galaxy->Bodies[bodyIndex + bm])/(MyTree->Nodes[curIndex + nm] + Galaxy->Bodies[bodyIndex + bm]);
  MyTree->Nodes[curIndex + nmy] = (MyTree->Nodes[curIndex + nmy] * MyTree->Nodes[curIndex + nm] + Galaxy->Bodies[bodyIndex + by] * Galaxy->Bodies[bodyIndex + bm])/(MyTree->Nodes[curIndex + nm] + Galaxy->Bodies[bodyIndex + bm]);
  MyTree->Nodes[curIndex + nm] += Galaxy->Bodies[bodyIndex + bm];
  

  //here im gonna do a recursion to inser the body further to the child
  if(curLevel < TREELEVEL){
    key = 0;
    if(Galaxy->Bodies[bodyIndex + bx] > (xmi + xma)/2) key += 1;
    if(Galaxy->Bodies[bodyIndex + by] > (ymi + yma)/2) key += 2;

    insertBody(bodyIndex, curIndex*4 + key+1, curLevel+1);

  }

}

//calculate the force of bodies
void forceBody(int bodyIndex, int curIndex, int curLevel){
  int i,len,num;
  long int bx,by,bm,bvx,bvy,bfx,bfy;
  long int nminx,nmaxx,nminy,nmaxy,nm,nmx,nmy,nch1,nch2,nch3,nch4;
  long int xmi,xma,ymi,yma;
  data_t x_dist, y_dist, dtsq, dist, nsize, force, m1, m2;

  len = Galaxy->len;
  num = MyTree->numNode;
//parameters for bodies
  bx = 0;
  by = len;
  bm = len * 2;
  bvx = len * 3;
  bvy = len * 4;
  bfx = len * 5;
  bfy = len * 6;
//parameters for treenodes
  nminx = 0;
  nmaxx = num;
  nminy = num*2;
  nmaxy = num*3;
  nm = num*4;
  nmx = num*5;
  nmy = num*6;
  nch1 = num*7;
  nch2 = num*8;
  nch3 = num*9;
  nch3 = num*10;

//node's xmin xmax ymin ymax
  xmi = MyTree->Nodes[curIndex + nminx];
  xma = MyTree->Nodes[curIndex + nmaxx];
  ymi = MyTree->Nodes[curIndex + nminy];
  yma = MyTree->Nodes[curIndex + nmaxy];
//calculate the distance
  x_dist = Galaxy->Bodies[bodyIndex + bx] - MyTree->Nodes[curIndex + nmx];
  y_dist = Galaxy->Bodies[bodyIndex + by] - MyTree->Nodes[curIndex + nmy];
  dtsq = x_dist*x_dist + y_dist*y_dist;
  dist = sqrt(dtsq);
  nsize = MyTree->Nodes[curIndex + nmaxx] - MyTree->Nodes[curIndex + nminx];


//calculation of force
  if(nsize/dist < THETA ){// case1, far enough
    if( dtsq > 1000 ){
      m1 = Galaxy->Bodies[bodyIndex + bm];
      m2 = MyTree->Nodes[curIndex + nm];
      force = (m1 * m2  * GRAVITYCONST * 6E10) / (dtsq);
      Galaxy->Bodies[bodyIndex + bfx] += (-x_dist/dist)*force;//(x_dist/dist)*force;
      Galaxy->Bodies[bodyIndex + bfy] += (-y_dist/dist)*force;//(y_dist/dist)*force;
    }
  }
   
  else if(curLevel < TREELEVEL){//case2 , not far enough but can be splited
    forceBody(bodyIndex, curIndex*4+1, curLevel+1);
    forceBody(bodyIndex, curIndex*4+2, curLevel+1);
    forceBody(bodyIndex, curIndex*4+3, curLevel+1);
    forceBody(bodyIndex, curIndex*4+4, curLevel+1);
  }

  else if(nsize/dist >= THETA){//case3 , not far enough but already the smallest cell, here we calculate it by its centre of mass , but it should be done in brute-force...
    if( dtsq > 1000 ){

      m1 = Galaxy->Bodies[bodyIndex + bm];
      m2 = MyTree->Nodes[curIndex + nm];
      force = (m1 * m2  * GRAVITYCONST * 6E10) / (dtsq);
      Galaxy->Bodies[bodyIndex + bfx] += (x_dist/dist)*force;
      Galaxy->Bodies[bodyIndex + bfy] += (y_dist/dist)*force;

    }
  }

}

/**************************************/
//the quad tree func for cpu barnes-hut
//construct tree, insert to tree, calculate the force and velocity and next position of bodies
void quadTree(){

  void reset_tree();
  reset_tree();// we reset the tree at every new time interval

  int i,j,len,num;
  len = Galaxy->len;
  num = MyTree->numNode;

  long int bx,by,bm,bvx,bvy,bfx,bfy;

  
  bx = 0;
  by = len;
  bm = len * 2;
  bvx = len * 3;
  bvy = len * 4;
  bfx = len * 5;
  bfy = len * 6;




  //insert bodies to the tree
  for(i = 0; i < len; i++){
    insertBody(i,0,1);
  }

  //compute force
  for(j = 0; j < len; j++){
    Galaxy->Bodies[j + bfx] = 0;
    Galaxy->Bodies[j + bfy] = 0;
    forceBody(j,0,1);
  }


  //apply forces
  for( i = 0 ; i < len ; i++){

    Galaxy->Bodies[i + bvx] += Galaxy->Bodies[i + bfx] * TIMESTEP / Galaxy->Bodies[i + bm]; //set vx += fx /m
    Galaxy->Bodies[i + bvy] += Galaxy->Bodies[i + bfy] * TIMESTEP / Galaxy->Bodies[i + bm]; //set vy += fy /m

    Galaxy->Bodies[i + bx] += Galaxy->Bodies[i + bvx] * TIMESTEP; //new x +=vx
    Galaxy->Bodies[i + by] += Galaxy->Bodies[i + bvy] * TIMESTEP; //new y +=vy

  }

}